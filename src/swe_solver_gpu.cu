#include "hip/hip_runtime.h"
//swe_solver_gpu.cu
#include "swe_solver.h"
#include <hip/hip_runtime.h>
#include <chrono>
#include <iostream>

__global__ void updateGPUKernel(double* d_eta, double* d_u, int grid_size) {
    int i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i >= 0 && i < grid_size) {
        double k1 = 0.5 * d_eta[i] + d_u[i];
        d_eta[i] += k1;
    }
}

void SWESolver::runGPU(int steps) {
    double *d_eta, *d_u;
    size_t size = grid_size * sizeof(double);

    hipMalloc((void**)&d_eta, size);
    hipMalloc((void**)&d_u, size);

    hipMemcpy(d_eta, eta.data(), size, hipMemcpyHostToDevice);
    hipMemcpy(d_u, u.data(), size, hipMemcpyHostToDevice);

    int threadsPerBlock = 1024;
    int blocksPerGrid = (grid_size + threadsPerBlock - 1) / threadsPerBlock;

    auto start = std::chrono::high_resolution_clock::now();
    for (int t = 0; t < steps; ++t) {
        updateGPUKernel<<<blocksPerGrid, threadsPerBlock>>>(d_eta, d_u, grid_size);
    }
    hipDeviceSynchronize();
    auto end = std::chrono::high_resolution_clock::now();
    
    execution_time = end - start;
    hipMemcpy(eta.data(), d_eta, size, hipMemcpyDeviceToHost);

    hipFree(d_eta);
    hipFree(d_u);

    printExecutionTime();
}
